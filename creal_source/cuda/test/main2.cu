
#include <hip/hip_runtime.h>
#include <stdio.h> //standard library
#include "/usr/include/linux/cuda.h" //cuda library
#include <stdlib.h>	
#include <unistd.h>

typedef unsigned int uint;

__global__ void kernel(int* A, int* B, int* C)
{
	
			int i = threadIdx.x; //this is used as a thread identifier 
			
			C[i] = A[i] + B[i];
	
}




/*more things need to be done right here 
 * you have to cast malloc in order to pass some stuff
 * ridiculous stuff going on
 * 
 * 
 * 
 * 
 * 
 * 	
 * */
 
void addNumber(int* var, int count)
{
	for (int $i = 0; $i < count; $i++)
	
	{
		var[$i] = 5;
		
	}
	
}
void zeroArray(int* var, int count)
{
	for (int $i = 0; $i < count; $i++)
	
	{
		var[$i] = 0;
		
	}
	
}

int sumArray(int* var, int count)
{
	int summation = 0;
	
	for (int $i = 0; $i < count; $i++)
	
	{
		summation += var[$i];
		
	}
	
	return summation;
	
}

int main (void)
{
	
	int N = 256; //N constant
	int sum; //the sum
	int* hostMem0 = (int*)malloc(sizeof(int) * N); //allocate one element of host memory
	int* hostMem1 = (int*)malloc(sizeof(int) * N); //allocate one element of host memory
	int* hostMem2 = (int*)malloc(sizeof(int) * N); //allocate one element of host memory
	
	
	
	
	addNumber(hostMem0,N);
	addNumber(hostMem1,N);
	zeroArray(hostMem2,N);
	
	
	
	
	int* cudaMem; //allocate one element of device memory
	
	hipMalloc(&cudaMem, sizeof(int) * N); //this will allocate memory inside of the cuda device
    
	//~ hostMem[0] = 0; //just assign the host memory value to zero
    //~ cudaMemcpy(cudaMem,hostMem,sizeof(int) * N,cudaMemcpyHostToDevice); //copy memory from host to the device
			sum = sumArray(hostMem2,N);
			printf("Value %d\n",sum);
		
			dim3 threads(N,N);
			
			kernel<<<1,threads>>>(hostMem0,hostMem1,cudaMem); //call the cuda kernel function
			
			
			printf("Sleeping for 2 \n"); //sleeping
			sleep(2); //sleep for 5 seconds
			
			
			hipMemcpy(hostMem2,cudaMem,sizeof(int) * N,hipMemcpyDeviceToHost); //copy memory from device to the host
			
			
			sum = sumArray(hostMem2,N);
			
			printf("Value %d\n\n",(uint)sum); //the new value
			
			//~ cudaMemcpy(cudaMem,hostMem,sizeof(int) * 1,cudaMemcpyHostToDevice); //copy memory from host to the device
		
			
			
				free(hostMem0); //free the host memory
				free(hostMem1); //free the host memory
				free(hostMem2); //free the host memory
				
				hipFree(cudaMem); //free device memory
		
    
				return 0; //return 0
		

}

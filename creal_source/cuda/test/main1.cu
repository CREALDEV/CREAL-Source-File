

#include <hip/hip_runtime.h>
#include <stdio.h> //standard library
#include "/usr/include/linux/cuda.h" //cuda library
#include <stdlib.h>	
#include <unistd.h>

__global__ void kernel(int* A)
{
	
	A[0]++;
	
}

//~ __global__ void vecAdd(int* A, int* B)
//~ {
	//~ 
	//~ 
//~ }



//~ __global__ void passTheTorch(int* A)
//~ {
	//~ 
		//~ int i = threadIdx.x; //this is used as a thread identifier 
		//~ 
//~ }


/*more things need to be done right here 
 * you have to cast malloc in order to pass some stuff
 * ridiculous stuff going on
 * 
 * 
 * 
 * 
 * 
 * 	
 * */
 
 
int main (void)
{
	
	
	int* hostMem = (int*)malloc(sizeof(int) * 1); //allocate one element of host memory
	int* cudaMem; //allocate one element of device memory
	hipMalloc(&cudaMem, sizeof(int) * 1); //this will allocate memory inside of the cuda device
    
	hostMem[0] = 0; //just assign the host memory value to zero
    hipMemcpy(cudaMem,hostMem,sizeof(int) * 1,hipMemcpyHostToDevice); //copy memory from host to the device
    
    
		for(int i = 0; i < 1000; i++) //iterate 1000 times
		
		{
			printf("Value %d\n",hostMem[0]);
    
			kernel<<<1,1>>>(cudaMem); //call the cuda kernel function
			hipMemcpy(hostMem,cudaMem,sizeof(int) * 1,hipMemcpyDeviceToHost); //copy memory from device to the host
			
			printf("Sleeping for 2 \n"); //sleeping
			sleep(2); //sleep for 5 seconds
			printf("Value %d\n\n",hostMem[0]); //the new value
			
			hipMemcpy(cudaMem,hostMem,sizeof(int) * 1,hipMemcpyHostToDevice); //copy memory from host to the device
		
		}
			
				free(hostMem); //free the host memory
				hipFree(cudaMem); //free device memory
		
    
    return 0; //return 0
		

}

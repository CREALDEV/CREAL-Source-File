#include "hip/hip_runtime.h"
/* Testing passing structures in cuda*/ 
//~ http://stackoverflow.com/questions/5670430/error-when-passing-pointer-to-struct-to-function-in-c  ~ this url has a nice solution

#include <stdio.h> //standard library
#include "/usr/include/linux/hip/hip_runtime.h" //cuda library
#include <stdlib.h>	
#include <unistd.h>
#include <string.h>
#include <time.h>
//~ #include "simplePrintf/cuPrintf.cu" //including the cuda simple printf stuff

#include "libcudacreal.h" //a json parser that is fast and efficient


//~ char const ADD = 0x01;
//~ char const DEL = 0x02;

//~ char const MOV = 0x03;
//~ char const MUL = 0x04;
//~ char const IOU = 0x05;


	 //~ struct node
	 //~ {
		 //~ 
		 //~ 
		//~ char *name; //this is the name of the node
		//~ int value; //this is the value that will be changed
		//~ uint index; //this is the index of the array that will be changed
		//~ char directive; // this is used to initiate a certain routine

		//~ 
		 //~ 
	 //~ };
	 //~ 
		//~ typedef struct node pre_crea_node; 
		//~ typedef pre_crea_node *crea_node;


	crea_node returnCreaNodeMemHost(uint num)
	{
		
		
		
		
		//remember you have to cast malloc with cuda	
		//~ return (crea_node)malloc(sizeof(pre_crea_node) * num); //make sure you allocate
		return (crea_node) calloc(num, sizeof(pre_crea_node));	
	}

	struct preNodeContainer
	{
		uint count; // this is the number of nodes
		crea_node creaNodeArray;
	};
		
	
	

// below is the main kernel for the program

	__global__ void kernel(crea_node creaNodeStructure)
		{

				int thrI = threadIdx.x; //this is used as a thread identifier 	
				int i = blockIdx.x * blockDim.x + threadIdx.x;
				
				int dI = creaNodeStructure[thrI].index;
				
				if(creaNodeStructure[i].directive == ADD) //this is for adding
					{
						
					creaNodeStructure[dI].value++;	
					//~ cuPrintf("Add Recieved\n");
						
						
					
				
					}
				
				if(creaNodeStructure[i].directive == DEL) //this is for deletion
					{
						creaNodeStructure[dI].value--;	
						
						
						//~ cuPrintf("Delete Recieved\n");
					
					
					}
					__syncthreads();
					//~ 
				//~ cuPrintf("Current Thread: %d\n", creaNodeStructure->value);
				
				//~ creaNodeStructure[thrI].value = 0;
				
				
		
		}



void freeNodeMemories(crea_node hostNode, crea_node deviceNode)
{
			free(hostNode); //free the host memory	
			hipFree(deviceNode); //free the device memory
}


int main (void)
{
					
	/* General app stuff */				
	//~ int errorTrap; //this is used for checking errors
	

		//~ cudaPrintfInit(); //initializes the cuda printf function
		int N = 0;
	
		
		crea_node hostNode = creatCreaNodes("jsonTest.json",&N);
		crea_node deviceNode;
		
		

		
		//~ 
		//~ populateCreaNodeStructure(&hostNodeStructure,&deviceNodeStructure,N); //this adds elements to the nodes
		
		hipMalloc(&deviceNode, sizeof(pre_crea_node) * N);
		
		//~ hipMemcpy(&deviceNodeStructure,&hostNodeStructure,sizeof(hostNodeStructure) * 1,hipMemcpyHostToDevice); //copy memory from host to the device
		
		hipMemcpy(deviceNode,hostNode,sizeof(pre_crea_node) * N,hipMemcpyHostToDevice); //copy memory from host to the device
		printf("%d",N);
		
		puts("\n\n");	
		
		
		for (int i = 0; i < N; i++)
		{
			
			printf("%d\n",hostNode[i].value);
			
			
		
		
		}
			
			puts("\n\n");	
		
		
		//~ processCreaNodes<<<1,N>>>(deviceNodeStructure);
		kernel<<<1,N>>>(deviceNode);
		
		//~ cudaPrintfDisplay(stdout, true); //displays the cuda printf functionality
		
		//~ hipMemcpy(&hostNodeStructure,&deviceNodeStructure,sizeof(hostNodeStructure) * 1,hipMemcpyDeviceToHost); //copy memory from device to the host
		
		//~ cudaPrintfEnd(); //ends the cuda printf function
		
		hipMemcpy(hostNode,deviceNode,sizeof(pre_crea_node) * N,hipMemcpyDeviceToHost); //copy memory from device to the host	
		
		
		for (int i = 0; i < N; i++)
		{
			
			printf("%d\n",hostNode[i].value);
			
			
		
		
		}
		
			
	
			//~ /*this is for testing purposes */	
	
			//~ for (int i = 0; i < hostNodeStructure.count; i++)
			//~ {
				//~ printf("%d\n",hostNodeStructure.creaNodeArray[i].value);
				//~ 
			//~ }
			//~ 
			
		
			
					
			
			//~ free(hostNode);
			freeNodeMemories(hostNode,deviceNode);
			
			return 0; //return 0 to the OS if everything was alright 
		
}


	

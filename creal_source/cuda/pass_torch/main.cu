#include "hip/hip_runtime.h"
#include <stdio.h> //standard library
#include "/usr/include/linux/hip/hip_runtime.h" //cuda library
#include "simplePrintf/cuPrintf.cu" //including the cuda simple printf stuff
#include <stdlib.h>	 //standard lib
#include <unistd.h> //universal standard

#define BLOCK_SIZE = 16 //constant
//just use this above as a constant to a thread block


__global__ void printThreadStuff()
{
	
			//~ int iThreadID = threadIdx.x; //this is used as a thread identifier 
			int nTotalThreads = blockDim.x;	// Total number of active threads
			unsigned int ThreadIDX= threadIdx.x+blockDim.x*blockIdx.x;
			
			
			__shared__ int sharedArray[1000]; //a shared value
			 
			
				
				
		
				sharedArray[0] = 0; //zero out the value
				
				
				
				for(; sharedArray[nTotalThreads - 1] < 1000000000;)
				{
		
					for (int i = 1; i < nTotalThreads; i++)
					{
						if (ThreadIDX == i)
						{
							sharedArray[ThreadIDX] = sharedArray[ThreadIDX - 1] + 1;
						
						}
						
						__syncthreads(); //syncs the threads up
					}
						sharedArray[0] = sharedArray[nTotalThreads - 1]; //zero out the value
						__syncthreads(); //syncs the threads up
				}		
				
				
				
				if(ThreadIDX == 0) //only use the first thread
				{
					
					cuPrintf("Total passed thread: %d\n",sharedArray[nTotalThreads - 1] );
				}
}


		/*more things need to be done right here 
		 * you have to c111ast malloc in order to pass some stuff
		 * ridiculous stuff going on
		 * 
		 * __shared__ float max[256]; // add this somewhere in new code base.. this is for a shared memory object
		 * 
		 * int nTotalThreads = blockDim.x;	// Total number of active threads
		 * __syncthreads(); //find out more about this function
		 * 	
		 * */




		typedef unsigned int uint; //unsigned


		int main(int argc, char **argv)
		{
		

			if ( argc != 2 )
			{
				
				
					//~ argc is actually whats on the command line..
				
					printf("Sorry %s \n", argv[0]); //show errata for the program
				
					
					
				
			}

		//else run the rest of the program
		
		else
		{
			
				//BEGIN//
				
					cudaPrintfInit(); //initializes the cuda printf function
			
					//~ int* tmpVar = (int*)malloc(sizeof(int) * 1); //allocate one element of host memory
					//~ tmpVar = (int*)argv[1]; //cast the argument as a integer pointer
					//~ printf("Doing this %s\n", argv[1] ); // for will print out the first argument 
		
					printThreadStuff<<<1,10>>>(); //call the cuda kernel function	
				
					cudaPrintfDisplay(stdout, true); //displays the cuda printf functionality
			
					//~ free(tmpVar); // free a temporary variable
				
				
				//END//
				
					cudaPrintfEnd(); //ends the cuda printf function
				
				
				return 0; //return 0
				
				
		}


}
	
